#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include<map>
#include<algorithm>
#include<chrono>
#include<time.h>
#include "hip/hip_runtime.h"
#include ""

#define MAXSTACKSIZE 8 //max stack size
#define MAXHSIZE 80000
#define MAXSSIZE 100000

#define MAXBATCHLEVELSIZE 128
#define MAXSBATCHSIZE 4096
#define MAXHBATCHSIZE 2048

#define BLOCKNUM 128
// #define BLOCKNUM 128
#define THREADNUM 64

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)


template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
	if (err != hipSuccess) {
		std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
		std::cerr << hipGetErrorString(err) << " " << func << std::endl;
		exit(1);
	}
}

class Vertex {
public:
	unsigned label;
	std::vector<unsigned> neighbor;
	Vertex() {}
	Vertex(unsigned lb) {
		label = lb;
	}
};

class CSR {
public:
	unsigned* row_offset = nullptr;
	unsigned* column_index = nullptr;
	CSR() {
		row_offset = nullptr;
		column_index = nullptr;
	}
	~CSR() {
		delete[] row_offset;
		delete[] column_index;
	}
};

class Graph {
public:
	std::vector<Vertex> vertices;
	unsigned vertex_num;
	unsigned edge_num;
	unsigned vertex_num_after_trim;
	unsigned edge_num_after_trim;
	CSR* csr;

	Graph() {
		vertex_num = 0;
		edge_num = 0;
		csr = NULL;
		vertex_num_after_trim = 0;
		edge_num_after_trim = 0;
	}

	void addVertex(unsigned lb);
	void addBipartiteEdge(unsigned lb1, unsigned lb2);
	void addEdge(unsigned lb1, unsigned lb2);
	void printGraph();
	void transformToCSR(CSR& _csr);
};

void
Graph::addVertex(unsigned lb) {
	vertices.push_back(Vertex(lb));
	vertex_num++;
}

void
Graph::addEdge(unsigned lb1, unsigned lb2) {
	vertices[lb1 - 1].neighbor.push_back(lb2);
	vertices[lb2 - 1].neighbor.push_back(lb1);
	edge_num++;
}

void
Graph::addBipartiteEdge(unsigned lb1, unsigned lb2) {
	vertices[lb1 - 1].neighbor.push_back(lb2);
	edge_num++;
}

void
Graph::printGraph() {
	std::cout << "Number of vertices: " << this->vertex_num << std::endl;
}

void
Graph::transformToCSR(CSR& _csr) {
	unsigned offset_size = this->vertex_num + 1;
	_csr.row_offset = new unsigned[offset_size];
	_csr.row_offset[0] = 0;
	unsigned sum = 0;
	for (int i = 1; i < offset_size; i++) {
		sum += this->vertices[i - 1].neighbor.size();
		_csr.row_offset[i] = sum;
	}
	//sum += this->vertices[offset_size - 1].neighbor.size();

	_csr.column_index = new unsigned[sum];
	unsigned k = 0;
	for (int i = 0; i < offset_size - 1; i++) {
		for (int j = 0; j < this->vertices[i].neighbor.size(); j++) {
			_csr.column_index[k] = this->vertices[i].neighbor[j];
			k++;
		}
	}
	this->csr = &_csr;
}

void readFile(Graph& graph, bool flag, char* path) {
	FILE* fp = NULL;
	fp = fopen(path, "r");
	int L_vertex_num, R_vertex_num, edge_num;
	fscanf(fp, "%d %d %d\n", &L_vertex_num, &R_vertex_num, &edge_num);
	// if flag is true, input the left layer
	if (flag == true) {
		for (unsigned i = 1; i <= L_vertex_num; i++) {
			graph.addVertex(i);
		}
		for (int j = 0; j < edge_num; j++) {
			unsigned in, out;
			fscanf(fp, "%u %u\n", &in, &out);
			graph.addBipartiteEdge(in, out);
		}
	}
	else {
		for (unsigned i = 1; i <= R_vertex_num; i++) {
			graph.addVertex(i);
		}
		for (int j = 0; j < edge_num; j++) {
			unsigned in, out;
			fscanf(fp, "%u %u\n", &in, &out);
			graph.addBipartiteEdge(out, in);
		}
	}
	fclose(fp);
}

void Collect2Hop(Graph& L, Graph& R, Graph& H, int q) {
	//for L
	unsigned L_num = L.vertex_num;
	for (unsigned m = 1; m <= L_num; m++) {
		H.addVertex(m);
	}
	for (unsigned i = 0; i < L_num; i++) {
		if (L.vertices[i].label != 0) {
			unsigned* list2hop = new unsigned[L_num]();
			int i_neighbor_num = L.vertices[i].neighbor.size();
			for (int j = 0; j < i_neighbor_num; j++) {
				unsigned j_vertex = L.vertices[i].neighbor[j];
				int j_neighbor_num = R.vertices[j_vertex - 1].neighbor.size();
				for (int k = 0; k < j_neighbor_num; k++) {
					list2hop[R.vertices[j_vertex - 1].neighbor[k] - 1]++;
				}
			}
			for (unsigned l = i + 1; l < L_num; l++) {
				if (list2hop[l] >= q) {
					H.addEdge(i + 1, l + 1);
				}
			}
			delete[] list2hop;
		}
	}
}

int BinarySearch(std::vector<unsigned>& nums, int size, unsigned t) {
	int left = 0, right = size - 1;
	while (left <= right) {
		int mid = left + ((right - left) / 2);
		if (nums[mid] > t) {
			right = mid - 1;
		}
		else if (nums[mid] < t) {
			left = mid + 1;
		}
		else {
			return mid;
		}
	}
	return -1;
}

bool cmp(std::pair<unsigned, unsigned> a, std::pair<unsigned, unsigned> b) {
	return a.second < b.second;
}

void edgeDirectingByDegree(Graph& H) {
	std::vector<std::pair<unsigned, unsigned>> lb_degree;
	unsigned vertex_num = H.vertex_num;
	//int count = 0;
	for (unsigned i = 0; i < vertex_num; i++) {
		lb_degree.push_back(std::pair<unsigned, unsigned>(i + 1, H.vertices[i].neighbor.size()));
	}
	sort(lb_degree.begin(), lb_degree.end(), cmp);
	/*for (int j = vertex_num - 1; j >= 0; j--) {
		for (int k = j - 1; k >= 0; k--) {
			std::vector<unsigned>::iterator find_val = find(H.vertices[lb_degree[j].first - 1].neighbor.begin(), H.vertices[lb_degree[j].first - 1].neighbor.end(), lb_degree[k].first);
			if (find_val != H.vertices[lb_degree[j].first - 1].neighbor.end()) {
				H.vertices[lb_degree[j].first - 1].neighbor.erase(find_val);
				count1++;
			}
		}
	}
	std::cout << "Deleted:" << count1 << std::endl;*/
	for (int j = 0; j < vertex_num; j++) {
		for (auto val : H.vertices[lb_degree[j].first - 1].neighbor) {
			// std::vector<unsigned>::iterator find_val = find(H.vertices[val - 1].neighbor.begin(), H.vertices[val - 1].neighbor.end(), lb_degree[j].first);
			// if (find_val != H.vertices[val - 1].neighbor.end()) {
			// 	H.vertices[val - 1].neighbor.erase(find_val);
			// 	//count++;
			// }
            int res = BinarySearch(H.vertices[val - 1].neighbor, H.vertices[val - 1].neighbor.size(), lb_degree[j].first);
			if (res != -1){
				std::vector<unsigned>::iterator find_val = H.vertices[val - 1].neighbor.begin() + res;
				H.vertices[val - 1].neighbor.erase(find_val);
			}
		}
	}
	//std::cout << "Deleted:" << count << std::endl;
}

void edgeDirectingByDegreeNew(Graph& H) {
	std::map<unsigned, unsigned> lb_degree;
	unsigned vertex_num = H.vertex_num;
	//int count = 0;
	for (unsigned i = 0; i < vertex_num; i++) {
		lb_degree[i + 1] =  H.vertices[i].neighbor.size();
	}
	Graph New_H;
	for (unsigned m = 1; m <= H.vertex_num; m++) {
		New_H.addVertex(m);
	}
	for (unsigned i = 0; i < vertex_num; i++) {
		if (i % 10000 == 0) printf("%d\n", i);
		for (unsigned val : H.vertices[i].neighbor) {
			if (lb_degree[i + 1] < lb_degree[val]) {
				New_H.vertices[i].neighbor.push_back(val);
				New_H.edge_num++;
			}
			else if (lb_degree[i + 1] == lb_degree[val]) {
				if (New_H.vertices[val - 1].neighbor.size() == 0) {
					New_H.vertices[i].neighbor.push_back(val);
					New_H.edge_num++;
				}
			}
		}
	}
	H = New_H;
}

void TrimGraphByCoreNew(Graph& L, Graph& R, int p, int q) {
	int count = 0, old_count = -1;
	unsigned right_removed_edge = 0, left_removed_edge = 0;
	std::vector<unsigned> left_removed, right_removed;
	//std::vector<Vertex>tmp_left, tmp_right;
	while (count != old_count) {
		old_count = count;
		right_removed_edge = 0, left_removed_edge = 0;
		left_removed.clear();
		right_removed.clear();
		//Process left layer
		for (unsigned i = 0; i < L.vertex_num; i++) {
			if (L.vertices[i].label != 0 && L.vertices[i].neighbor.size() < q) {
				left_removed.push_back(i + 1);
				L.vertices[i].label = 0;
				count++;
			}
		}

		L.vertex_num_after_trim = L.vertex_num_after_trim == 0 ? L.vertex_num - left_removed.size() : L.vertex_num_after_trim - left_removed.size();

		//for (int j = 0; j < R.vertex_num; j++) {
		for (int i = 0; i < left_removed.size(); i++){
			unsigned re_vertex = left_removed[i];
			for (int j = 0; j < L.vertices[re_vertex - 1].neighbor.size(); j++) {
				unsigned vertex = L.vertices[re_vertex - 1].neighbor[j];
				int it = BinarySearch(R.vertices[vertex - 1].neighbor, R.vertices[vertex - 1].neighbor.size(), re_vertex);
				if (it != -1) {
					R.vertices[vertex - 1].neighbor.erase(R.vertices[vertex - 1].neighbor.begin() + it);
					left_removed_edge++;
				}
			}
			L.vertices[re_vertex - 1].neighbor.clear();
		}
		//printf("Now Left: %d\n", L.vertex_num_after_trim);

		L.edge_num_after_trim = L.edge_num_after_trim == 0 ? L.edge_num - left_removed_edge : L.edge_num_after_trim - left_removed_edge;
		R.edge_num_after_trim = R.edge_num_after_trim == 0 ? R.edge_num - left_removed_edge : R.edge_num_after_trim - left_removed_edge;
		//Right layer
		for (unsigned i = 0; i < R.vertex_num; i++) {
			if (R.vertices[i].label != 0 && R.vertices[i].neighbor.size() < p) {
				right_removed.push_back(i + 1);
				R.vertices[i].label = 0;
				count++;
			}
		}

		R.vertex_num_after_trim = R.vertex_num_after_trim == 0 ? R.vertex_num - right_removed.size() : R.vertex_num_after_trim - right_removed.size();

		for (int i = 0; i < right_removed.size(); i++) {
			unsigned re_vertex = right_removed[i];
			for (int j = 0; j < R.vertices[re_vertex - 1].neighbor.size(); j++) {
				unsigned vertex = R.vertices[re_vertex - 1].neighbor[j];
				//auto it = find(L.vertices[j].neighbor.begin(), L.vertices[j].neighbor.end(), right_removed[i]);
				int it = BinarySearch(L.vertices[vertex - 1].neighbor, L.vertices[vertex - 1].neighbor.size(), re_vertex);
				if (it != -1) {
					L.vertices[vertex - 1].neighbor.erase(L.vertices[vertex - 1].neighbor.begin() + it);
					right_removed_edge++;
				}
			}
			R.vertices[re_vertex - 1].neighbor.clear();
		}
		//printf("Now Right: %d\n", R.vertex_num_after_trim);

		R.edge_num_after_trim = R.edge_num_after_trim == 0 ? R.edge_num - right_removed_edge : R.edge_num_after_trim - right_removed_edge;
		L.edge_num_after_trim = L.edge_num_after_trim == 0 ? L.edge_num - right_removed_edge : L.edge_num_after_trim - right_removed_edge;
	}
}

void reformatGraph(Graph& L, Graph& R) {
	std::map<unsigned, unsigned> dic_L, dic_R;
	unsigned count_L = 1, count_R = 1;
	for (unsigned i = 0; i < L.vertex_num; i++) {
		if (L.vertices[i].label != 0) {
			dic_L[i + 1] = count_L;
			count_L++;
		}
	}
	for (unsigned i = 0; i < R.vertex_num; i++) {
		if (R.vertices[i].label != 0) {
			dic_R[i + 1] = count_R;
			count_R++;
		}
	}
	// printf("%d %d\n", count_L - 1, count_R - 1);
	count_L = 0, count_R = 0;
	for (unsigned i = 0; i < L.vertex_num; i++) {
		for (unsigned j = 0; j < L.vertices[i].neighbor.size(); j++) {
			L.vertices[i].neighbor[j] = dic_R[L.vertices[i].neighbor[j]];
			//count_L++;
		}
	}
	for (unsigned i = 0; i < R.vertex_num; i++) {
		for (unsigned j = 0; j < R.vertices[i].neighbor.size(); j++) {
			R.vertices[i].neighbor[j] = dic_L[R.vertices[i].neighbor[j]];
			//count_L++;
		}
	}
	for (unsigned i = 0; i < L.vertex_num; i++) {
		if (L.vertices[i].label != 0) {
			L.vertices[count_L] = L.vertices[i];
			count_L++;
		}
	}
	for (unsigned i = 0; i < R.vertex_num; i++) {
		if (R.vertices[i].label != 0) {
			R.vertices[count_R] = R.vertices[i];
			count_R++;
		}
	}
	// printf("%d %d\n", count_L, count_R);
	L.vertex_num = L.vertex_num_after_trim;
	R.vertex_num = R.vertex_num_after_trim;
	L.edge_num = L.edge_num_after_trim;
	R.edge_num = R.edge_num_after_trim;
	dic_L.clear();
	dic_R.clear();
}

__device__ unsigned stack[BLOCKNUM][MAXSTACKSIZE];
__device__ unsigned subH[BLOCKNUM][MAXSTACKSIZE][MAXHSIZE] = { 0 };
__device__ unsigned S[BLOCKNUM][MAXSTACKSIZE][MAXSSIZE] = { 0 };

__device__ unsigned offset_H[BLOCKNUM][MAXSTACKSIZE][MAXHSIZE] = { 0 };
__device__ unsigned offset_L[BLOCKNUM][MAXSTACKSIZE][MAXSSIZE] = { 0 };

__device__ unsigned glockArray[BLOCKNUM] = { 0 };

__device__ unsigned batch_info[BLOCKNUM][MAXSTACKSIZE][3] = { 0 }; // 0 is batch index, 1 is batch size, 2 is next_k
__device__ unsigned GCL[BLOCKNUM] = { 0 };

__device__  unsigned long long OrderMulDev(unsigned m, unsigned n) {
	// if (n == 0 || n == m) {
	// 	return 1;
	// }
	// return OrderMulDev(m - 1, n) + OrderMulDev(m - 1, n - 1);
	unsigned long long ans = 1;
    for(unsigned i = 1; i <= n; i++){
        ans *= m--;
		ans /= i; 
    }
    return ans;
}

__device__ void IntersectionDevOld2(unsigned* B, int B_size, unsigned A[], unsigned Res_size, unsigned Res[]) {
	int tid_t = threadIdx.x < 32 ? threadIdx.x : threadIdx.x - 32;
	while (tid_t < Res_size) {
		unsigned val = A[tid_t];
		//binary search
		int ret = 0, tmp = B_size;
		while (tmp > 1) {
			int halfsize = tmp / 2;
			int cand = B[ret + halfsize];
			ret += (cand < val) ? halfsize : 0;
			tmp -= halfsize;
		}
		ret += (B[ret] < val);
		Res[tid_t] = (ret <= (B_size - 1) ? (B[ret] == val) ? val : 0 : 0);
		tid_t += (blockDim.x / 2);
	}
}

__device__ void IntersectionDevOld3(unsigned* B, int B_size, unsigned A[], unsigned Res_size) {
	int tid_t = threadIdx.x < 32 ? threadIdx.x : threadIdx.x - 32;
	while (tid_t < Res_size) {
		unsigned val = A[tid_t];
		//binary search
		int ret = 0, tmp = B_size;
		while (tmp > 1) {
			int halfsize = tmp / 2;
			int cand = B[ret + halfsize];
			ret += (cand < val) ? halfsize : 0;
			tmp -= halfsize;
		}
		ret += (B[ret] < val);
		// A[tid_t] = (ret <= (B_size - 1) ? (B[ret] == val) ? val : 0 : 0);
		if (ret > B_size - 1 || B[ret] != val) A[tid_t] = 0;
		tid_t += (blockDim.x / 2);
	}
}

__device__ void IntersectionDev6( unsigned* A, unsigned* begin, unsigned* end, unsigned* Res, unsigned Res_num, int size) {
	int tid_t = threadIdx.x < 32 ? threadIdx.x : threadIdx.x - 32;
	while (tid_t < Res_num * size) {
		int idx = tid_t / Res_num;
		int A_size = end[idx] - begin[idx];
		unsigned val = Res[tid_t];
		int ret = 0, tmp = A_size;
		while (tmp > 1) {
			int halfsize = tmp / 2;
			int cand = A[begin[idx] + ret + halfsize];
			ret += (cand < val) ? halfsize : 0;
			tmp -= halfsize;
		}
		ret += (A[begin[idx] + ret] < val);
		Res[tid_t] = (ret <= (A_size - 1) ? (A[begin[idx] + ret] == val) ? val : 0 : 0);
		tid_t += (blockDim.x / 2);
	}
}

__global__ void findCliqueGPUNew7Dis2LevelStealWorkBatch(unsigned* row_dev_L, unsigned* col_dev_L, unsigned* row_dev_H, unsigned* col_dev_H, unsigned long long* count, unsigned* p, unsigned* q, unsigned* Hsize, unsigned* non_vertex, unsigned* non_vertex_father) {
	__shared__ int top, level;
	__shared__ unsigned next_k, tid, batch_size;
	__shared__ unsigned begin_L[MAXBATCHLEVELSIZE], end_L[MAXBATCHLEVELSIZE], begin_H[MAXBATCHLEVELSIZE], end_H[MAXBATCHLEVELSIZE], Res_S[MAXSBATCHSIZE], Res_H[MAXHBATCHSIZE], Num_L[MAXSTACKSIZE][MAXBATCHLEVELSIZE], Num_H[MAXSTACKSIZE][MAXBATCHLEVELSIZE];
	__shared__ unsigned next_node_id, skip, min_gcl, idx_gcl;
    tid = blockIdx.x;
	__syncthreads();
	while (tid < *Hsize) {
		__syncthreads();
		if (threadIdx.x == 0) {
			top = 0;
			level = 0;
			next_k = 0;
            glockArray[blockIdx.x] = 1;
			stack[blockIdx.x][top] = 0;
			printf("%d:%d\n", blockIdx.x, tid);
		}
		__syncthreads();
		if (threadIdx.x < 32) {
			unsigned num_S = row_dev_L[non_vertex[tid] + 1] - row_dev_L[non_vertex[tid]], num_L = row_dev_L[non_vertex_father[tid] + 1] - row_dev_L[non_vertex_father[tid]];
			if (num_S < num_L) {
				unsigned batch_begin_S = num_S / MAXSBATCHSIZE;
				unsigned last_batch = num_S - batch_begin_S * MAXSBATCHSIZE;
				if (threadIdx.x == 0) {
					Num_L[level][0] = 0;
					S[blockIdx.x][level][0] = 1;
				}
				for (int i = 0; i < batch_begin_S; i++) {
					// 填充Res_S
					for (int j = threadIdx.x; j < MAXSBATCHSIZE; j += (THREADNUM / 2)) {
						Res_S[j] = col_dev_L[row_dev_L[non_vertex[tid]] + j + i * MAXSBATCHSIZE];
					}
					// 计算该Batch的交集
					IntersectionDevOld3(&col_dev_L[row_dev_L[non_vertex_father[tid]]], num_L, Res_S, MAXSBATCHSIZE);
					if (threadIdx.x == 0) {
						for (int k = 0; k < MAXSBATCHSIZE; k++) {
							if (Res_S[k] != 0) S[blockIdx.x][level][++Num_L[level][0]] = Res_S[k];
						}
					}
				}
				// 计算最后一个batch
				for (int j = threadIdx.x; j < last_batch; j += (THREADNUM / 2)) {
					Res_S[j] = col_dev_L[row_dev_L[non_vertex[tid]] + j + batch_begin_S * MAXSBATCHSIZE];
				}
				IntersectionDevOld3(&col_dev_L[row_dev_L[non_vertex_father[tid]]], num_L, Res_S, last_batch);
				if (threadIdx.x == 0) {
					for (int k = 0; k < last_batch; k++) {
						if (Res_S[k] != 0) S[blockIdx.x][level][++Num_L[level][0]] = Res_S[k];
					}
				}
			}
			else {
				// if(threadIdx.x == 0) if(num_L > 6000) printf("Num_L: %d\n", num_L);
				// IntersectionDevOld2(&col_dev_L[row_dev_L[non_vertex[tid]]], num_S, &col_dev_L[row_dev_L[non_vertex_father[tid]]], num_L, Res_S);
				unsigned batch_begin_L = num_L / MAXSBATCHSIZE;
				unsigned last_batch = num_L - batch_begin_L * MAXSBATCHSIZE;
				if (threadIdx.x == 0) {
					Num_L[level][0] = 0;
					S[blockIdx.x][level][0] = 1;
				}
				for (int i = 0; i < batch_begin_L; i++) {
					// 填充Res_S
					for (int j = threadIdx.x; j < MAXSBATCHSIZE; j += (THREADNUM / 2)) {
						Res_S[j] = col_dev_L[row_dev_L[non_vertex_father[tid]] + j + i * MAXSBATCHSIZE];
					}
					// 计算该Batch的交集
					IntersectionDevOld3(&col_dev_L[row_dev_L[non_vertex[tid]]], num_S, Res_S, MAXSBATCHSIZE);
					if (threadIdx.x == 0) {
						for (int k = 0; k < MAXSBATCHSIZE; k++) {
							if (Res_S[k] != 0) S[blockIdx.x][level][++Num_L[level][0]] = Res_S[k];
						}
					}
				}
				// 计算最后一个batch
				for (int j = threadIdx.x; j < last_batch; j += (THREADNUM / 2)) {
					Res_S[j] = col_dev_L[row_dev_L[non_vertex_father[tid]] + j + batch_begin_L * MAXSBATCHSIZE];
				}
				IntersectionDevOld3(&col_dev_L[row_dev_L[non_vertex[tid]]], num_S, Res_S, last_batch);
				if (threadIdx.x == 0) {
					for (int k = 0; k < last_batch; k++) {
						if (Res_S[k] != 0) S[blockIdx.x][level][++Num_L[level][0]] = Res_S[k];
					}
				}
			}
		}
		else {
			unsigned num_subH = row_dev_H[non_vertex[tid] + 1] - row_dev_H[non_vertex[tid]], num_H = row_dev_H[non_vertex_father[tid] + 1] - row_dev_H[non_vertex_father[tid]];
			if (num_subH < num_H) {
				// IntersectionDevOld2(&col_dev_H[row_dev_H[non_vertex_father[tid]]], num_H, &col_dev_H[row_dev_H[non_vertex[tid]]], num_subH, Res_H);
				unsigned batch_begin_subH = num_subH / MAXHBATCHSIZE;
				unsigned last_batch = num_subH - batch_begin_subH * MAXHBATCHSIZE;
				if (threadIdx.x == 32) {
					Num_H[level][0] = 0;
					subH[blockIdx.x][level][0] = 1;
				}
				for (int i = 0; i < batch_begin_subH; i++) {
					// 填充Res_S
					for (int j = threadIdx.x - 32; j < MAXHBATCHSIZE; j += (THREADNUM / 2)) {
						Res_H[j] = col_dev_H[row_dev_H[non_vertex[tid]] + j + i * MAXHBATCHSIZE];
					}
					// 计算该Batch的交集
					IntersectionDevOld3(&col_dev_H[row_dev_H[non_vertex_father[tid]]], num_H, Res_H, MAXHBATCHSIZE);
					if (threadIdx.x == 32) {
						for (int k = 0; k < MAXHBATCHSIZE; k++) {
							if (Res_H[k] != 0) subH[blockIdx.x][level][++Num_H[level][0]] = Res_H[k];
						}
					}
				}
				// 计算最后一个batch
				for (int j = threadIdx.x - 32; j < last_batch; j += (THREADNUM / 2)) {
					Res_H[j] = col_dev_H[row_dev_H[non_vertex[tid]] + j + batch_begin_subH * MAXHBATCHSIZE];
				}
				IntersectionDevOld3(&col_dev_H[row_dev_H[non_vertex_father[tid]]], num_H, Res_H, last_batch);
				if (threadIdx.x == 32) {
					for (int k = 0; k < last_batch; k++) {
						if (Res_H[k] != 0) subH[blockIdx.x][level][++Num_H[level][0]] = Res_H[k];
					}
				}
			}
			else {
				// IntersectionDevOld2(&col_dev_H[row_dev_H[non_vertex[tid]]], num_subH, &col_dev_H[row_dev_H[non_vertex_father[tid]]], num_H, Res_H);
				unsigned batch_begin_H = num_H / MAXHBATCHSIZE;
				unsigned last_batch = num_H - batch_begin_H * MAXHBATCHSIZE;
				if (threadIdx.x == 32) {
					Num_H[level][0] = 0;
					subH[blockIdx.x][level][0] = 1;
				}
				for (int i = 0; i < batch_begin_H; i++) {
					// 填充Res_S
					for (int j = threadIdx.x - 32; j < MAXHBATCHSIZE; j += (THREADNUM / 2)) {
						Res_H[j] = col_dev_H[row_dev_H[non_vertex_father[tid]] + j + i * MAXHBATCHSIZE];
					}
					// 计算该Batch的交集
					IntersectionDevOld3(&col_dev_H[row_dev_H[non_vertex[tid]]], num_subH, Res_H, MAXHBATCHSIZE);
					if (threadIdx.x == 32) {
						for (int k = 0; k < MAXHBATCHSIZE; k++) {
							if (Res_H[k] != 0) subH[blockIdx.x][level][++Num_H[level][0]] = Res_H[k];
						}
					}
				}
				// 计算最后一个batch
				for (int j = threadIdx.x - 32; j < last_batch; j += (THREADNUM / 2)) {
					Res_H[j] = col_dev_H[row_dev_H[non_vertex_father[tid]] + j + batch_begin_H * MAXHBATCHSIZE];
				}
				IntersectionDevOld3(&col_dev_H[row_dev_H[non_vertex[tid]]], num_subH, Res_H, last_batch);
				if (threadIdx.x == 32) {
					for (int k = 0; k < last_batch; k++) {
						if (Res_H[k] != 0) subH[blockIdx.x][level][++Num_H[level][0]] = Res_H[k];
					}
				}
			}
		}
		__syncthreads();
        if (threadIdx.x == 0){
		    if (Num_L[level][0] >= *q && Num_H[level][0] >= *p - level - 2) {
				level++;
				top++;
			}
			// __syncthreads();
		}
		__syncthreads();
		while (top != 0) {
			__syncthreads();
			unsigned batch_idx = batch_info[blockIdx.x][level][0];
            __syncthreads();
			if (threadIdx.x == 0) {
				if (batch_idx == 0) {
					int size = Num_H[level - 1][next_k];
					S[blockIdx.x][level][0] = size;
					subH[blockIdx.x][level][0] = size;
					batch_info[blockIdx.x][level][1] = size / MAXBATCHLEVELSIZE;
				}
				unsigned all_batch = batch_info[blockIdx.x][level][1];
				batch_size = (batch_idx < all_batch ? MAXBATCHLEVELSIZE : subH[blockIdx.x][level][0] - all_batch * MAXBATCHLEVELSIZE);
			}
			__syncthreads();
			for (int j = threadIdx.x; j < batch_size; j += THREADNUM) {
				unsigned vertex_tmp = subH[blockIdx.x][level - 1][offset_H[blockIdx.x][level - 1][batch_info[blockIdx.x][level - 1][2]] + j + 1 + MAXBATCHLEVELSIZE * batch_idx] - 1;

				end_L[j] = row_dev_L[vertex_tmp + 1];
				begin_L[j] = row_dev_L[vertex_tmp];

				end_H[j] = row_dev_H[vertex_tmp + 1];
				begin_H[j] = row_dev_H[vertex_tmp];
			}
			__syncthreads();
			if (threadIdx.x < 32) {
				unsigned inter_offset = batch_info[blockIdx.x][level - 1][2] + 1;
				unsigned inter_idx = offset_L[blockIdx.x][level - 1][inter_offset - 1];
				unsigned* begin_S = &S[blockIdx.x][level - 1][inter_idx + 1];
				unsigned num_S = Num_L[level - 1][inter_offset - 1];
				for (int i = threadIdx.x; i < batch_size; i += (THREADNUM / 2)) {
					offset_L[blockIdx.x][level][i] = i * num_S;
				}
				if (batch_size * num_S < MAXSBATCHSIZE) {
					// if(threadIdx.x == 0) if(num_S * size > 6000) printf("Size: %d, Num_S: %d, Size*Num_S: %d\n", size, num_S, size*num_S);
					// 将S扩至size倍
					for (int i = 0; i < batch_size; i++) {
						// if (threadIdx.x < num_S) {
						// 	Res_S[threadIdx.x + i * num_S] = begin_S[threadIdx.x];
						// }
                        for(int j = threadIdx.x; j < num_S; j += (THREADNUM / 2)){
                            Res_S[j + i * num_S] = begin_S[j];
                        }
					}
					IntersectionDev6(col_dev_L, begin_L, end_L, Res_S, num_S, batch_size);
					for (int i = threadIdx.x; i < batch_size; i += (THREADNUM / 2)) {
						int res_offset = i * num_S;
						Num_L[level][i] = 0;
						for (int k = 0; k < num_S; k++) {
							if (Res_S[k + res_offset] != 0) {
								S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k + res_offset];
							}
						}
					}
				}
				else {
					for (int i = 0; i < batch_size; i++) {
						// IntersectionDevOld2(&col_dev_L[begin_L[i]], end_L[i] - begin_L[i], begin_S, num_S, Res_S);
						// if (threadIdx.x == 0) {
						// 	int res_offset = i * num_S;
						// 	Num_L[level][i] = 0;
						// 	for (int k = 0; k < num_S; k++) {
						// 		if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
						// 	}
						// }
                        unsigned num_L = end_L[i] - begin_L[i];
                        if (num_S < num_L) {
                            unsigned batch_begin_S = num_S / MAXSBATCHSIZE;
                            unsigned last_batch = num_S - batch_begin_S * MAXSBATCHSIZE;
                            if (threadIdx.x == 0) {
                                Num_L[level][i] = 0;
                                // S[blockIdx.x][level][0] = 1;
                            }
                            for (int m = 0; m < batch_begin_S; m++) {
                                // 填充Res_S
                                for (int j = threadIdx.x; j < MAXSBATCHSIZE; j += (THREADNUM / 2)) {
                                    // Res_S[j] = col_dev_L[row_dev_L[non_vertex[tid]] + j + i * MAXSBATCHSIZE];
                                    Res_S[j] = begin_S[j + m * MAXSBATCHSIZE];
                                }
                                // 计算该Batch的交集
                                IntersectionDevOld3(&col_dev_L[begin_L[i]], num_L, Res_S, MAXSBATCHSIZE);
                                if (threadIdx.x == 0) {
                                    int res_offset = i * num_S;
                                    for (int k = 0; k < MAXSBATCHSIZE; k++) {
                                        if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                                    }
                                }
                            }
                            // 计算最后一个batch
                            for (int j = threadIdx.x; j < last_batch; j += (THREADNUM / 2)) {
                                // Res_S[j] = col_dev_L[row_dev_L[non_vertex[tid]] + j + batch_begin_S];
                                Res_S[j] = begin_S[j + batch_begin_S * MAXSBATCHSIZE];
                            }
                            IntersectionDevOld3(&col_dev_L[begin_L[i]], num_L, Res_S, last_batch);
                            if (threadIdx.x == 0) {
                                int res_offset = i * num_S;
                                for (int k = 0; k < last_batch; k++) {
                                    if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                                }
                            }
                        }
                        else {
                            unsigned batch_begin_L = num_L / MAXSBATCHSIZE;
                            unsigned last_batch = num_L - batch_begin_L * MAXSBATCHSIZE;
                            if (threadIdx.x == 0) {
                                Num_L[level][i] = 0;
                                // S[blockIdx.x][level][0] = 1;
                            }
                            for (int m = 0; m < batch_begin_L; m++) {
                                // 填充Res_S
                                for (int j = threadIdx.x; j < MAXSBATCHSIZE; j += (THREADNUM / 2)) {
                                    Res_S[j] = col_dev_L[begin_L[i] + j + m * MAXSBATCHSIZE];
                                }
                                // 计算该Batch的交集
                                IntersectionDevOld3(begin_S, num_S, Res_S, MAXSBATCHSIZE);
                                if (threadIdx.x == 0) {
                                    int res_offset = i * num_S;
                                    for (int k = 0; k < MAXSBATCHSIZE; k++) {
                                        if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                                    }
                                }
                            }
                            // 计算最后一个batch
                            for (int j = threadIdx.x; j < last_batch; j += (THREADNUM / 2)) {
                                Res_S[j] = col_dev_L[begin_L[i] + j + batch_begin_L * MAXSBATCHSIZE];
                            }
                            IntersectionDevOld3(begin_S, num_S, Res_S, last_batch);
                            if (threadIdx.x == 0) {
                                int res_offset = i * num_S;
                                for (int k = 0; k < last_batch; k++) {
                                    if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                                }
                            }
                        }
					}
				}
			}
			else {
				unsigned inter_offset = batch_info[blockIdx.x][level - 1][2] + 1;
				unsigned inter_idx = offset_H[blockIdx.x][level - 1][inter_offset - 1];
				unsigned* begin_subH = &subH[blockIdx.x][level - 1][inter_idx + 1];
				unsigned num_subH = Num_H[level - 1][inter_offset - 1];
				for (int i = threadIdx.x - 32; i < batch_size; i += (THREADNUM / 2)) {
					offset_H[blockIdx.x][level][i] = i * num_subH;
				}
				if (batch_size * num_subH < MAXHBATCHSIZE) {
					// 将subH扩至size倍
					for (int i = 0; i < batch_size; i++) {
						// if (threadIdx.x - 32 < num_subH) {
						// 	Res_H[threadIdx.x - 32 + i * num_subH] = begin_subH[threadIdx.x - 32];
						// }
                        for(int j = threadIdx.x - 32; j < num_subH; j += (THREADNUM / 2)){
                            Res_H[j + i * num_subH] = begin_subH[j];
                        }
					}
					IntersectionDev6(col_dev_H, begin_H, end_H, Res_H, num_subH, batch_size);
					for (int i = threadIdx.x - 32; i < batch_size; i += (THREADNUM / 2)) {
						int res_offset = i * num_subH;
						Num_H[level][i] = 0;
						for (int k = 0; k < num_subH; k++) {
							if (Res_H[k + res_offset] != 0) {
								subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k + res_offset];
							}
						}
					}
				}
				else {
					for (int i = 0; i < batch_size; i++) {
						// IntersectionDevOld2(&col_dev_H[begin_H[i]], end_H[i] - begin_H[i], begin_subH, num_subH, Res_H);
						// if (threadIdx.x == 32) {
						// 	int res_offset = i * num_subH;
						// 	Num_H[level][i] = 0;
						// 	for (int k = 0; k < num_subH; k++) {
						// 		if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
						// 	}
						// }
                        unsigned num_H = end_H[i] - begin_H[i];
                        if (num_subH < num_H) {
                            // IntersectionDevOld2(&col_dev_H[row_dev_H[non_vertex_father[tid]]], num_H, &col_dev_H[row_dev_H[non_vertex[tid]]], num_subH, Res_H);
                            unsigned batch_begin_subH = num_subH / MAXHBATCHSIZE;
                            unsigned last_batch = num_subH - batch_begin_subH * MAXHBATCHSIZE;
                            if (threadIdx.x == 32) {
                                Num_H[level][i] = 0;
                                // subH[blockIdx.x][level][0] = 1;
                            }
                            for (int m = 0; m < batch_begin_subH; m++) {
                                // 填充Res_S
                                for (int j = threadIdx.x - 32; j < MAXHBATCHSIZE; j += (THREADNUM / 2)) {
                                    Res_H[j] = begin_subH[j + m * MAXHBATCHSIZE];
                                }
                                // 计算该Batch的交集
                                IntersectionDevOld3(&col_dev_H[begin_H[i]], num_H, Res_H, MAXHBATCHSIZE);
                                if (threadIdx.x == 32) {
                                    int res_offset = i * num_subH;
                                    for (int k = 0; k < MAXHBATCHSIZE; k++) {
                                        if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                                    }
                                }
                            }
                            // 计算最后一个batch
                            for (int j = threadIdx.x - 32; j < last_batch; j += (THREADNUM / 2)) {
                                Res_H[j] = begin_subH[j + batch_begin_subH * MAXHBATCHSIZE];
                            }
                            IntersectionDevOld3(&col_dev_H[begin_H[i]], num_H, Res_H, last_batch);
                            if (threadIdx.x == 32) {
                                int res_offset = i * num_subH;
                                for (int k = 0; k < last_batch; k++) {
                                    if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                                }
                            }
                        }
                        else {
                            // IntersectionDevOld2(&col_dev_H[row_dev_H[non_vertex[tid]]], num_subH, &col_dev_H[row_dev_H[non_vertex_father[tid]]], num_H, Res_H);
                            unsigned batch_begin_H = num_H / MAXHBATCHSIZE;
                            unsigned last_batch = num_H - batch_begin_H * MAXHBATCHSIZE;
                            if (threadIdx.x == 32) {
                                Num_H[level][i] = 0;
                                // subH[blockIdx.x][level][0] = 1;
                            }
                            for (int m = 0; m < batch_begin_H; m++) {
                                // 填充Res_S
                                for (int j = threadIdx.x - 32; j < MAXHBATCHSIZE; j += (THREADNUM / 2)) {
                                    Res_H[j] = col_dev_H[begin_H[i] + j + m * MAXHBATCHSIZE];
                                }
                                // 计算该Batch的交集
                                IntersectionDevOld3(begin_subH, num_subH, Res_H, MAXHBATCHSIZE);
                                if (threadIdx.x == 32) {
                                    int res_offset = i * num_subH;
                                    for (int k = 0; k < MAXHBATCHSIZE; k++) {
                                        if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                                    }
                                }
                            }
                            // 计算最后一个batch
                            for (int j = threadIdx.x - 32; j < last_batch; j += (THREADNUM / 2)) {
                                Res_H[j] = col_dev_H[begin_H[i] + j + batch_begin_H * MAXHBATCHSIZE];
                            }
                            IntersectionDevOld3(begin_subH, num_subH, Res_H, last_batch);
                            if (threadIdx.x == 32) {
                                int res_offset = i * num_subH;
                                for (int k = 0; k < last_batch; k++) {
                                    if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                                }
                            }
                        }
					}
				}
			}
			__syncthreads();
			if (level == *p - 2) {
				__syncthreads();
				for (int i = threadIdx.x; i < batch_size; i += blockDim.x) {
					unsigned m = Num_L[level][i];
					if (m >= *q) {
						atomicAdd(count, OrderMulDev(m, *q));
					}
				}
				__syncthreads();
				if (threadIdx.x == 0) {
					stack[blockIdx.x][top] = 0;
				}
				__syncthreads();
				if (batch_info[blockIdx.x][level][0] == batch_info[blockIdx.x][level][1]) {
					if (threadIdx.x == 0) {
						batch_info[blockIdx.x][level][0] = 0;
                        batch_info[blockIdx.x][level][1] = 0;
						batch_info[blockIdx.x][level][2] = 0;
						top--;
						level--;
						unsigned all_batch_tmp = batch_info[blockIdx.x][level][1];
						batch_size = (batch_info[blockIdx.x][level][0] < all_batch_tmp ? MAXBATCHLEVELSIZE : subH[blockIdx.x][level][0] - all_batch_tmp * MAXBATCHLEVELSIZE);
					}
                    __syncthreads();
				}
				else {
					if (threadIdx.x == 0) {
						batch_info[blockIdx.x][level][0]++;
					}
					__syncthreads();
					continue;
				}
				// __syncthreads();
			}
			__syncthreads();
			if (threadIdx.x == 0) {
				while (top != 0) {
					int flag = 0;
					for (int k = stack[blockIdx.x][level]; k < batch_size; k++) {
						if (Num_L[level][k] >= *q && Num_H[level][k] >= *p - level - 2) {
							stack[blockIdx.x][level] = k + 1;
							next_k = k;
							batch_info[blockIdx.x][level][2] = k;
							flag = 1;
							top++;
							level++;
							break;
						}
					}
					if (flag == 0) {
						stack[blockIdx.x][level] = 0;
						if (batch_info[blockIdx.x][level][0] == batch_info[blockIdx.x][level][1]) {
							batch_info[blockIdx.x][level][0] = 0;
                            batch_info[blockIdx.x][level][1] = 0;
						    batch_info[blockIdx.x][level][2] = 0;
							level--;
							top--;
							unsigned all_batch_tmp = batch_info[blockIdx.x][level][1];
							batch_size = (batch_info[blockIdx.x][level][0] < all_batch_tmp ? MAXBATCHLEVELSIZE : subH[blockIdx.x][level][0] - all_batch_tmp * MAXBATCHLEVELSIZE);
						}
						else {
							batch_info[blockIdx.x][level][0]++;
							break;
						}
					}
					else {
						break;
					}
				}
			}
			__syncthreads();
		}
        __syncthreads();
		if (threadIdx.x == 0) {
			while (atomicExch(&glockArray[blockIdx.x], 0) == 0);
			// while (atomicCAS(&glockArray[blockIdx.x], 0, 1) != 0);
			unsigned tmp_gcl_ori = GCL[blockIdx.x];
			if (tmp_gcl_ori != 0xFFFFFFFF) {
				tid = (tmp_gcl_ori + 1) * BLOCKNUM + blockIdx.x;
				// printf("block:%d, GCL:%d\n", blockIdx.x, tmp_gcl_ori);
				// atomicAdd(&GCL[blockIdx.x], 1);
                GCL[blockIdx.x]++;
			}
			else {
				tid = 0xFFFFFFFF;
			}
			glockArray[blockIdx.x] = 1;
            // atomicExch(&glockArray[blockIdx.x], 1);
		}
		__syncthreads();
	}
    __syncthreads();
	if (threadIdx.x == 0) {
		//printf("%d end %f s, %f s\n", blockIdx.x, (float)duration / 1000000, (float)duration1 / 1000000);
		// atomicExch(&GCL[blockIdx.x], 0xFFFFFFFF);
        GCL[blockIdx.x] = 0xFFFFFFFF;
		// printf("block %d starts to steal work\n", blockIdx.x);
	}
	// if(threadIdx.x == 0 || threadIdx.x == 32) printf("A block:%d, thread:%d\n",blockIdx.x,threadIdx.x);
	
	__syncthreads();
	while (true) {
		__syncthreads();
		if (threadIdx.x == 0) {
			min_gcl = 0xFFFFFFFF, idx_gcl = 0xFFFFFFFF, skip = 0;
			for (int i = ((blockIdx.x + 1) % BLOCKNUM); i != blockIdx.x; i = (i + 1) % BLOCKNUM) {
				unsigned tmp = GCL[i];
				if (tmp != 0xFFFFFFFF) {
					min_gcl = tmp;
					idx_gcl = i;
					break;
				}
			}
			printf("B");
		}
		__syncthreads();
		if (min_gcl == 0xFFFFFFFF) {
			__syncthreads();
			break;
		}
		__syncthreads();
		if(threadIdx.x == 0) {
			while (atomicExch(&glockArray[idx_gcl], 0) == 0);
            // while (atomicCAS(&glockArray[idx_gcl], 0, 1) != 0);
			next_node_id = (min_gcl + 1) * BLOCKNUM + idx_gcl;
			if (GCL[idx_gcl] != min_gcl) {
				skip = 1;
                // printf("A");
			}
			if (next_node_id >= *Hsize) {
				skip = 1;
				// atomicExch(&GCL[idx_gcl], 0xFFFFFFFF);
                GCL[idx_gcl] = 0xFFFFFFFF;
                // printf("A");
			}
			if (skip == 0) {
				atomicAdd(&GCL[idx_gcl], 1);
                // printf("A");
                // printf("B");
			}

			glockArray[idx_gcl] = 1;
            // atomicExch(&glockArray[idx_gcl], 1);
		}
		__syncthreads();
		if (skip == 0) {
            __syncthreads();
            if (threadIdx.x == 0) {
                top = 0;
                level = 0;
                next_k = 0;
                stack[blockIdx.x][top] = 0;
                printf("%d Steal %d's %d\n", blockIdx.x, idx_gcl, next_node_id);
            }
            __syncthreads();
            if (threadIdx.x < 32) {
                unsigned num_S = row_dev_L[non_vertex[next_node_id] + 1] - row_dev_L[non_vertex[next_node_id]], num_L = row_dev_L[non_vertex_father[next_node_id] + 1] - row_dev_L[non_vertex_father[next_node_id]];
                if (num_S < num_L) {
                    unsigned batch_begin_S = num_S / MAXSBATCHSIZE;
                    unsigned last_batch = num_S - batch_begin_S * MAXSBATCHSIZE;
                    if (threadIdx.x == 0) {
                        Num_L[level][0] = 0;
                        S[blockIdx.x][level][0] = 1;
                    }
                    for (int i = 0; i < batch_begin_S; i++) {
                        // 填充Res_S
                        for (int j = threadIdx.x; j < MAXSBATCHSIZE; j += (THREADNUM / 2)) {
                            Res_S[j] = col_dev_L[row_dev_L[non_vertex[next_node_id]] + j + i * MAXSBATCHSIZE];
                        }
                        // 计算该Batch的交集
                        IntersectionDevOld3(&col_dev_L[row_dev_L[non_vertex_father[next_node_id]]], num_L, Res_S, MAXSBATCHSIZE);
                        if (threadIdx.x == 0) {
                            for (int k = 0; k < MAXSBATCHSIZE; k++) {
                                if (Res_S[k] != 0) S[blockIdx.x][level][++Num_L[level][0]] = Res_S[k];
                            }
                        }
                    }
                    // 计算最后一个batch
                    for (int j = threadIdx.x; j < last_batch; j += (THREADNUM / 2)) {
                        Res_S[j] = col_dev_L[row_dev_L[non_vertex[next_node_id]] + j + batch_begin_S * MAXSBATCHSIZE];
                    }
                    IntersectionDevOld3(&col_dev_L[row_dev_L[non_vertex_father[next_node_id]]], num_L, Res_S, last_batch);
                    if (threadIdx.x == 0) {
                        for (int k = 0; k < last_batch; k++) {
                            if (Res_S[k] != 0) S[blockIdx.x][level][++Num_L[level][0]] = Res_S[k];
                        }
                    }
                }
                else {
                    // if(threadIdx.x == 0) if(num_L > 6000) printf("Num_L: %d\n", num_L);
                    // IntersectionDevOld2(&col_dev_L[row_dev_L[non_vertex[next_node_id]]], num_S, &col_dev_L[row_dev_L[non_vertex_father[next_node_id]]], num_L, Res_S);
                    unsigned batch_begin_L = num_L / MAXSBATCHSIZE;
                    unsigned last_batch = num_L - batch_begin_L * MAXSBATCHSIZE;
                    if (threadIdx.x == 0) {
                        Num_L[level][0] = 0;
                        S[blockIdx.x][level][0] = 1;
                    }
                    for (int i = 0; i < batch_begin_L; i++) {
                        // 填充Res_S
                        for (int j = threadIdx.x; j < MAXSBATCHSIZE; j += (THREADNUM / 2)) {
                            Res_S[j] = col_dev_L[row_dev_L[non_vertex_father[next_node_id]] + j + i * MAXSBATCHSIZE];
                        }
                        // 计算该Batch的交集
                        IntersectionDevOld3(&col_dev_L[row_dev_L[non_vertex[next_node_id]]], num_S, Res_S, MAXSBATCHSIZE);
                        if (threadIdx.x == 0) {
                            for (int k = 0; k < MAXSBATCHSIZE; k++) {
                                if (Res_S[k] != 0) S[blockIdx.x][level][++Num_L[level][0]] = Res_S[k];
                            }
                        }
                    }
                    // 计算最后一个batch
                    for (int j = threadIdx.x; j < last_batch; j += (THREADNUM / 2)) {
                        Res_S[j] = col_dev_L[row_dev_L[non_vertex_father[next_node_id]] + j + batch_begin_L * MAXSBATCHSIZE];
                    }
                    IntersectionDevOld3(&col_dev_L[row_dev_L[non_vertex[next_node_id]]], num_S, Res_S, last_batch);
                    if (threadIdx.x == 0) {
                        for (int k = 0; k < last_batch; k++) {
                            if (Res_S[k] != 0) S[blockIdx.x][level][++Num_L[level][0]] = Res_S[k];
                        }
                    }
                }
            }
            else {
                unsigned num_subH = row_dev_H[non_vertex[next_node_id] + 1] - row_dev_H[non_vertex[next_node_id]], num_H = row_dev_H[non_vertex_father[next_node_id] + 1] - row_dev_H[non_vertex_father[next_node_id]];
                if (num_subH < num_H) {
                    // IntersectionDevOld2(&col_dev_H[row_dev_H[non_vertex_father[next_node_id]]], num_H, &col_dev_H[row_dev_H[non_vertex[next_node_id]]], num_subH, Res_H);
                    unsigned batch_begin_subH = num_subH / MAXHBATCHSIZE;
                    unsigned last_batch = num_subH - batch_begin_subH * MAXHBATCHSIZE;
                    if (threadIdx.x == 32) {
                        Num_H[level][0] = 0;
                        subH[blockIdx.x][level][0] = 1;
                    }
                    for (int i = 0; i < batch_begin_subH; i++) {
                        // 填充Res_S
                        for (int j = threadIdx.x - 32; j < MAXHBATCHSIZE; j += (THREADNUM / 2)) {
                            Res_H[j] = col_dev_H[row_dev_H[non_vertex[next_node_id]] + j + i * MAXHBATCHSIZE];
                        }
                        // 计算该Batch的交集
                        IntersectionDevOld3(&col_dev_H[row_dev_H[non_vertex_father[next_node_id]]], num_H, Res_H, MAXHBATCHSIZE);
                        if (threadIdx.x == 32) {
                            for (int k = 0; k < MAXHBATCHSIZE; k++) {
                                if (Res_H[k] != 0) subH[blockIdx.x][level][++Num_H[level][0]] = Res_H[k];
                            }
                        }
                    }
                    // 计算最后一个batch
                    for (int j = threadIdx.x - 32; j < last_batch; j += (THREADNUM / 2)) {
                        Res_H[j] = col_dev_H[row_dev_H[non_vertex[next_node_id]] + j + batch_begin_subH * MAXHBATCHSIZE];
                    }
                    IntersectionDevOld3(&col_dev_H[row_dev_H[non_vertex_father[next_node_id]]], num_H, Res_H, last_batch);
                    if (threadIdx.x == 32) {
                        for (int k = 0; k < last_batch; k++) {
                            if (Res_H[k] != 0) subH[blockIdx.x][level][++Num_H[level][0]] = Res_H[k];
                        }
                    }
                }
                else {
                    // IntersectionDevOld2(&col_dev_H[row_dev_H[non_vertex[next_node_id]]], num_subH, &col_dev_H[row_dev_H[non_vertex_father[next_node_id]]], num_H, Res_H);
                    unsigned batch_begin_H = num_H / MAXHBATCHSIZE;
                    unsigned last_batch = num_H - batch_begin_H * MAXHBATCHSIZE;
                    if (threadIdx.x == 32) {
                        Num_H[level][0] = 0;
                        subH[blockIdx.x][level][0] = 1;
                    }
                    for (int i = 0; i < batch_begin_H; i++) {
                        // 填充Res_S
                        for (int j = threadIdx.x - 32; j < MAXHBATCHSIZE; j += (THREADNUM / 2)) {
                            Res_H[j] = col_dev_H[row_dev_H[non_vertex_father[next_node_id]] + j + i * MAXHBATCHSIZE];
                        }
                        // 计算该Batch的交集
                        IntersectionDevOld3(&col_dev_H[row_dev_H[non_vertex[next_node_id]]], num_subH, Res_H, MAXHBATCHSIZE);
                        if (threadIdx.x == 32) {
                            for (int k = 0; k < MAXHBATCHSIZE; k++) {
                                if (Res_H[k] != 0) subH[blockIdx.x][level][++Num_H[level][0]] = Res_H[k];
                            }
                        }
                    }
                    // 计算最后一个batch
                    for (int j = threadIdx.x - 32; j < last_batch; j += (THREADNUM / 2)) {
                        Res_H[j] = col_dev_H[row_dev_H[non_vertex_father[next_node_id]] + j + batch_begin_H * MAXHBATCHSIZE];
                    }
                    IntersectionDevOld3(&col_dev_H[row_dev_H[non_vertex[next_node_id]]], num_subH, Res_H, last_batch);
                    if (threadIdx.x == 32) {
                        for (int k = 0; k < last_batch; k++) {
                            if (Res_H[k] != 0) subH[blockIdx.x][level][++Num_H[level][0]] = Res_H[k];
                        }
                    }
                }
            }
            __syncthreads();
            if (Num_L[level][0] >= *q && Num_H[level][0] >= *p - level - 2) {
                __syncthreads();
                if (threadIdx.x == 0) {
                    level++;
                    top++;
                }
                __syncthreads();
            }
            __syncthreads();
            while (top != 0) {
                __syncthreads();
                unsigned batch_idx = batch_info[blockIdx.x][level][0];
                __syncthreads();
                if (threadIdx.x == 0) {
                    if (batch_idx == 0) {
                        int size = Num_H[level - 1][next_k];
                        S[blockIdx.x][level][0] = size;
                        subH[blockIdx.x][level][0] = size;
                        batch_info[blockIdx.x][level][1] = size / MAXBATCHLEVELSIZE;
                    }
                    unsigned all_batch = batch_info[blockIdx.x][level][1];
                    batch_size = (batch_idx < all_batch ? MAXBATCHLEVELSIZE : subH[blockIdx.x][level][0] - all_batch * MAXBATCHLEVELSIZE);
                }
                __syncthreads();
                for (int j = threadIdx.x; j < batch_size; j += THREADNUM) {
                    unsigned vertex_tmp = subH[blockIdx.x][level - 1][offset_H[blockIdx.x][level - 1][batch_info[blockIdx.x][level - 1][2]] + j + 1 + MAXBATCHLEVELSIZE * batch_idx] - 1;

                    end_L[j] = row_dev_L[vertex_tmp + 1];
                    begin_L[j] = row_dev_L[vertex_tmp];

                    end_H[j] = row_dev_H[vertex_tmp + 1];
                    begin_H[j] = row_dev_H[vertex_tmp];
                }
                __syncthreads();
                if (threadIdx.x < 32) {
                    unsigned inter_offset = batch_info[blockIdx.x][level - 1][2] + 1;
                    unsigned inter_idx = offset_L[blockIdx.x][level - 1][inter_offset - 1];
                    unsigned* begin_S = &S[blockIdx.x][level - 1][inter_idx + 1];
                    unsigned num_S = Num_L[level - 1][inter_offset - 1];
                    for (int i = threadIdx.x; i < batch_size; i += (THREADNUM / 2)) {
                        offset_L[blockIdx.x][level][i] = i * num_S;
                    }
                    if (batch_size * num_S < MAXSBATCHSIZE) {
                        // if(threadIdx.x == 0) if(num_S * size > 6000) printf("Size: %d, Num_S: %d, Size*Num_S: %d\n", size, num_S, size*num_S);
                        // 将S扩至size倍
                        for (int i = 0; i < batch_size; i++) {
                            // if (threadIdx.x < num_S) {
                            // 	Res_S[threadIdx.x + i * num_S] = begin_S[threadIdx.x];
                            // }
                            for(int j = threadIdx.x; j < num_S; j += (THREADNUM / 2)){
                                Res_S[j + i * num_S] = begin_S[j];
                            }
                        }
                        IntersectionDev6(col_dev_L, begin_L, end_L, Res_S, num_S, batch_size);
                        for (int i = threadIdx.x; i < batch_size; i += (THREADNUM / 2)) {
                            int res_offset = i * num_S;
                            Num_L[level][i] = 0;
                            for (int k = 0; k < num_S; k++) {
                                if (Res_S[k + res_offset] != 0) {
                                    S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k + res_offset];
                                }
                            }
                        }
                    }
                    else {
                        for (int i = 0; i < batch_size; i++) {
                            // IntersectionDevOld2(&col_dev_L[begin_L[i]], end_L[i] - begin_L[i], begin_S, num_S, Res_S);
                            // if (threadIdx.x == 0) {
                            // 	int res_offset = i * num_S;
                            // 	Num_L[level][i] = 0;
                            // 	for (int k = 0; k < num_S; k++) {
                            // 		if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                            // 	}
                            // }
                            unsigned num_L = end_L[i] - begin_L[i];
                            if (num_S < num_L) {
                                unsigned batch_begin_S = num_S / MAXSBATCHSIZE;
                                unsigned last_batch = num_S - batch_begin_S * MAXSBATCHSIZE;
                                if (threadIdx.x == 0) {
                                    Num_L[level][i] = 0;
                                    // S[blockIdx.x][level][0] = 1;
                                }
                                for (int m = 0; m < batch_begin_S; m++) {
                                    // 填充Res_S
                                    for (int j = threadIdx.x; j < MAXSBATCHSIZE; j += (THREADNUM / 2)) {
                                        // Res_S[j] = col_dev_L[row_dev_L[non_vertex[tid]] + j + i * MAXSBATCHSIZE];
                                        Res_S[j] = begin_S[j + m * MAXSBATCHSIZE];
                                    }
                                    // 计算该Batch的交集
                                    IntersectionDevOld3(&col_dev_L[begin_L[i]], num_L, Res_S, MAXSBATCHSIZE);
                                    if (threadIdx.x == 0) {
                                        int res_offset = i * num_S;
                                        for (int k = 0; k < MAXSBATCHSIZE; k++) {
                                            if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                                        }
                                    }
                                }
                                // 计算最后一个batch
                                for (int j = threadIdx.x; j < last_batch; j += (THREADNUM / 2)) {
                                    // Res_S[j] = col_dev_L[row_dev_L[non_vertex[tid]] + j + batch_begin_S];
                                    Res_S[j] = begin_S[j + batch_begin_S * MAXSBATCHSIZE];
                                }
                                IntersectionDevOld3(&col_dev_L[begin_L[i]], num_L, Res_S, last_batch);
                                if (threadIdx.x == 0) {
                                    int res_offset = i * num_S;
                                    for (int k = 0; k < last_batch; k++) {
                                        if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                                    }
                                }
                            }
                            else {
                                unsigned batch_begin_L = num_L / MAXSBATCHSIZE;
                                unsigned last_batch = num_L - batch_begin_L * MAXSBATCHSIZE;
                                if (threadIdx.x == 0) {
                                    Num_L[level][i] = 0;
                                    // S[blockIdx.x][level][0] = 1;
                                }
                                for (int m = 0; m < batch_begin_L; m++) {
                                    // 填充Res_S
                                    for (int j = threadIdx.x; j < MAXSBATCHSIZE; j += (THREADNUM / 2)) {
                                        Res_S[j] = col_dev_L[begin_L[i] + j + m * MAXSBATCHSIZE];
                                    }
                                    // 计算该Batch的交集
                                    IntersectionDevOld3(begin_S, num_S, Res_S, MAXSBATCHSIZE);
                                    if (threadIdx.x == 0) {
                                        int res_offset = i * num_S;
                                        for (int k = 0; k < MAXSBATCHSIZE; k++) {
                                            if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                                        }
                                    }
                                }
                                // 计算最后一个batch
                                for (int j = threadIdx.x; j < last_batch; j += (THREADNUM / 2)) {
                                    Res_S[j] = col_dev_L[begin_L[i] + j + batch_begin_L * MAXSBATCHSIZE];
                                }
                                IntersectionDevOld3(begin_S, num_S, Res_S, last_batch);
                                if (threadIdx.x == 0) {
                                    int res_offset = i * num_S;
                                    for (int k = 0; k < last_batch; k++) {
                                        if (Res_S[k] != 0) S[blockIdx.x][level][res_offset + (++Num_L[level][i])] = Res_S[k];
                                    }
                                }
                            }
                        }
                    }
                }
                else {
                    unsigned inter_offset = batch_info[blockIdx.x][level - 1][2] + 1;
                    unsigned inter_idx = offset_H[blockIdx.x][level - 1][inter_offset - 1];
                    unsigned* begin_subH = &subH[blockIdx.x][level - 1][inter_idx + 1];
                    unsigned num_subH = Num_H[level - 1][inter_offset - 1];
                    for (int i = threadIdx.x - 32; i < batch_size; i += (THREADNUM / 2)) {
                        offset_H[blockIdx.x][level][i] = i * num_subH;
                    }
                    if (batch_size * num_subH < MAXHBATCHSIZE) {
                        // 将subH扩至size倍
                        for (int i = 0; i < batch_size; i++) {
                            // if (threadIdx.x - 32 < num_subH) {
                            // 	Res_H[threadIdx.x - 32 + i * num_subH] = begin_subH[threadIdx.x - 32];
                            // }
                            for(int j = threadIdx.x - 32; j < num_subH; j += (THREADNUM / 2)){
                                Res_H[j + i * num_subH] = begin_subH[j];
                            }
                        }
                        IntersectionDev6(col_dev_H, begin_H, end_H, Res_H, num_subH, batch_size);
                        for (int i = threadIdx.x - 32; i < batch_size; i += (THREADNUM / 2)) {
                            int res_offset = i * num_subH;
                            Num_H[level][i] = 0;
                            for (int k = 0; k < num_subH; k++) {
                                if (Res_H[k + res_offset] != 0) {
                                    subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k + res_offset];
                                }
                            }
                        }
                    }
                    else {
                        for (int i = 0; i < batch_size; i++) {
                            // IntersectionDevOld2(&col_dev_H[begin_H[i]], end_H[i] - begin_H[i], begin_subH, num_subH, Res_H);
                            // if (threadIdx.x == 32) {
                            // 	int res_offset = i * num_subH;
                            // 	Num_H[level][i] = 0;
                            // 	for (int k = 0; k < num_subH; k++) {
                            // 		if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                            // 	}
                            // }
                            unsigned num_H = end_H[i] - begin_H[i];
                            if (num_subH < num_H) {
                                // IntersectionDevOld2(&col_dev_H[row_dev_H[non_vertex_father[tid]]], num_H, &col_dev_H[row_dev_H[non_vertex[tid]]], num_subH, Res_H);
                                unsigned batch_begin_subH = num_subH / MAXHBATCHSIZE;
                                unsigned last_batch = num_subH - batch_begin_subH * MAXHBATCHSIZE;
                                if (threadIdx.x == 32) {
                                    Num_H[level][i] = 0;
                                    // subH[blockIdx.x][level][0] = 1;
                                }
                                for (int m = 0; m < batch_begin_subH; m++) {
                                    // 填充Res_S
                                    for (int j = threadIdx.x - 32; j < MAXHBATCHSIZE; j += (THREADNUM / 2)) {
                                        Res_H[j] = begin_subH[j + m * MAXHBATCHSIZE];
                                    }
                                    // 计算该Batch的交集
                                    IntersectionDevOld3(&col_dev_H[begin_H[i]], num_H, Res_H, MAXHBATCHSIZE);
                                    if (threadIdx.x == 32) {
                                        int res_offset = i * num_subH;
                                        for (int k = 0; k < MAXHBATCHSIZE; k++) {
                                            if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                                        }
                                    }
                                }
                                // 计算最后一个batch
                                for (int j = threadIdx.x - 32; j < last_batch; j += (THREADNUM / 2)) {
                                    Res_H[j] = begin_subH[j + batch_begin_subH * MAXHBATCHSIZE];
                                }
                                IntersectionDevOld3(&col_dev_H[begin_H[i]], num_H, Res_H, last_batch);
                                if (threadIdx.x == 32) {
                                    int res_offset = i * num_subH;
                                    for (int k = 0; k < last_batch; k++) {
                                        if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                                    }
                                }
                            }
                            else {
                                // IntersectionDevOld2(&col_dev_H[row_dev_H[non_vertex[tid]]], num_subH, &col_dev_H[row_dev_H[non_vertex_father[tid]]], num_H, Res_H);
                                unsigned batch_begin_H = num_H / MAXHBATCHSIZE;
                                unsigned last_batch = num_H - batch_begin_H * MAXHBATCHSIZE;
                                if (threadIdx.x == 32) {
                                    Num_H[level][i] = 0;
                                    // subH[blockIdx.x][level][0] = 1;
                                }
                                for (int m = 0; m < batch_begin_H; m++) {
                                    // 填充Res_S
                                    for (int j = threadIdx.x - 32; j < MAXHBATCHSIZE; j += (THREADNUM / 2)) {
                                        Res_H[j] = col_dev_H[begin_H[i] + j + m * MAXHBATCHSIZE];
                                    }
                                    // 计算该Batch的交集
                                    IntersectionDevOld3(begin_subH, num_subH, Res_H, MAXHBATCHSIZE);
                                    if (threadIdx.x == 32) {
                                        int res_offset = i * num_subH;
                                        for (int k = 0; k < MAXHBATCHSIZE; k++) {
                                            if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                                        }
                                    }
                                }
                                // 计算最后一个batch
                                for (int j = threadIdx.x - 32; j < last_batch; j += (THREADNUM / 2)) {
                                    Res_H[j] = col_dev_H[begin_H[i] + j + batch_begin_H * MAXHBATCHSIZE];
                                }
                                IntersectionDevOld3(begin_subH, num_subH, Res_H, last_batch);
                                if (threadIdx.x == 32) {
                                    int res_offset = i * num_subH;
                                    for (int k = 0; k < last_batch; k++) {
                                        if (Res_H[k] != 0) subH[blockIdx.x][level][res_offset + (++Num_H[level][i])] = Res_H[k];
                                    }
                                }
                            }
                        }
                    }
                }
                __syncthreads();
                if (level == *p - 2) {
                    __syncthreads();
                    for (int i = threadIdx.x; i < batch_size; i += blockDim.x) {
                        unsigned m = Num_L[level][i];
                        if (m >= *q) {
                            atomicAdd(count, OrderMulDev(m, *q));
                        }
                    }
                    __syncthreads();
                    if (threadIdx.x == 0) {
                        stack[blockIdx.x][top] = 0;
                    }
                    __syncthreads();
                    if (batch_info[blockIdx.x][level][0] == batch_info[blockIdx.x][level][1]) {
                        if (threadIdx.x == 0) {
                            batch_info[blockIdx.x][level][0] = 0;
                            batch_info[blockIdx.x][level][1] = 0;
                            batch_info[blockIdx.x][level][2] = 0;
                            top--;
                            level--;
                            unsigned all_batch_tmp = batch_info[blockIdx.x][level][1];
                            batch_size = (batch_info[blockIdx.x][level][0] < all_batch_tmp ? MAXBATCHLEVELSIZE : subH[blockIdx.x][level][0] - all_batch_tmp * MAXBATCHLEVELSIZE);
                        }
                        __syncthreads();
                    }
                    else {
                        if (threadIdx.x == 0) {
                            batch_info[blockIdx.x][level][0]++;
                        }
                        __syncthreads();
                        continue;
                    }
                    // __syncthreads();
                }
                __syncthreads();
                if (threadIdx.x == 0) {
                    while (top != 0) {
                        int flag = 0;
                        for (int k = stack[blockIdx.x][level]; k < batch_size; k++) {
                            if (Num_L[level][k] >= *q && Num_H[level][k] >= *p - level - 2) {
                                stack[blockIdx.x][level] = k + 1;
                                next_k = k;
                                batch_info[blockIdx.x][level][2] = k;
                                flag = 1;
                                top++;
                                level++;
                                break;
                            }
                        }
                        if (flag == 0) {
                            stack[blockIdx.x][level] = 0;
                            if (batch_info[blockIdx.x][level][0] == batch_info[blockIdx.x][level][1]) {
                                batch_info[blockIdx.x][level][0] = 0;
                                batch_info[blockIdx.x][level][1] = 0;
                                batch_info[blockIdx.x][level][2] = 0;
                                level--;
                                top--;
                                unsigned all_batch_tmp = batch_info[blockIdx.x][level][1];
                                batch_size = (batch_info[blockIdx.x][level][0] < all_batch_tmp ? MAXBATCHLEVELSIZE : subH[blockIdx.x][level][0] - all_batch_tmp * MAXBATCHLEVELSIZE);
                            }
                            else {
                                batch_info[blockIdx.x][level][0]++;
                                break;
                            }
                        }
                        else {
                            break;
                        }
                    }
                }
                __syncthreads();
            }
            __syncthreads();
        }
        __syncthreads();
    }
    __syncthreads();
}


int main(int argc, char* argv[]) {
    unsigned p = atoi(argv[2]), q = atoi(argv[3]);
    char* path = argv[1];
    CSR csrL;
    CSR csrH;
    Graph graphL;

    if(argc < 5){
        printf("Too few arguments\n");
        return 0;
    }
    else{
        printf("p: %s, q: %s, file: %s, left: %s\n", argv[2], argv[3], argv[1], argv[4]);
    }

    if (atoi(argv[4]) == 0) {
        printf("Select left to construct H\n");
        readFile(graphL, true, path);
        for (int i = 0; i < graphL.vertex_num; i++) {
            //printf("%d,", graphL.vertices[i].neighbor.size());
            sort(graphL.vertices[i].neighbor.begin(), graphL.vertices[i].neighbor.end());
        }
        std::cout << "vertexNum:" << graphL.vertex_num << "; edgeNum:" << graphL.edge_num << std::endl;


        Graph graphR;
        readFile(graphR, false, path);
        for (int i = 0; i < graphR.vertex_num; i++) {
            sort(graphR.vertices[i].neighbor.begin(), graphR.vertices[i].neighbor.end());
        }
        std::cout << "vertexNum:" << graphR.vertex_num << "; edgeNum:" << graphR.edge_num << std::endl;

        TrimGraphByCoreNew(graphL, graphR, p, q);
        std::cout << "vertexNum:" << graphL.vertex_num_after_trim << "; edgeNum:" << graphL.edge_num_after_trim << std::endl;
        std::cout << "vertexNum:" << graphR.vertex_num_after_trim << "; edgeNum:" << graphR.edge_num_after_trim << std::endl;

        // reformatGraph(graphL, graphR);
		// std::cout << "Reformat Done" << std::endl;

        Graph graphH;
        Collect2Hop(graphL, graphR, graphH, q);
        std::cout << "Construct H End" << std::endl;
        // std::cout << "vertexNum:" << graphH.vertex_num << "; edgeNum:" << graphH.edge_num << std::endl;
        edgeDirectingByDegreeNew(graphH);
        std::cout << "vertexNum:" << graphH.vertex_num << "; edgeNum:" << graphH.edge_num << std::endl;

        int zero_count = 0;
        std::vector<unsigned> nonzerovertex;
        for (int i = 0; i < graphH.vertex_num; i++) {
            if (graphH.vertices[i].neighbor.size() < p - 1) {
                zero_count++;
            }
            else {
                nonzerovertex.push_back(i);
            }
        }
        std::cout << "Number of degree-zero vertex: " << zero_count << "; Others: " << nonzerovertex.size() << std::endl;

        //******** New Stretagy ************
        std::vector<unsigned> firstlevel;
        std::vector<unsigned> firstlevelfather;
        for (int i = 0; i < nonzerovertex.size(); i++) {
            for (int j = 0; j < graphH.vertices[nonzerovertex[i]].neighbor.size(); j++) {
                firstlevel.push_back(graphH.vertices[nonzerovertex[i]].neighbor[j] - 1);
                firstlevelfather.push_back(nonzerovertex[i]);
            }
        }
        std::cout << "Number of first level: " << firstlevel.size() << std::endl;
        //**********************************

        /*unsigned aa = nonzerovertex[514];
        nonzerovertex.clear();
        nonzerovertex.push_back(aa);*/
        /*for (int i = 0; i < nonzerovertex.size(); i++) {
            printf("%d,", graphH.vertices[nonzerovertex[i]].neighbor.size());
        }*/

        /*unsigned* all_nei = new unsigned[12]{};
        for (int i = 0; i < nonzerovertex.size(); i++) {
            printf("tid:%d, num:%d, binum:%d\n", i, graphH.vertices[nonzerovertex[i]].neighbor.size(), graphL.vertices[nonzerovertex[i]].neighbor.size());
            all_nei[i % 12] += graphH.vertices[nonzerovertex[i]].neighbor.size();
        }
        for (int i = 0; i < 12; i++) printf("block: %d, allNeighbor:%d\n", i, all_nei[i]);*/

        graphL.transformToCSR(csrL);
        graphH.transformToCSR(csrH);

        //warm up GPU
        int* warmup = NULL;
        hipMalloc(&warmup, sizeof(int));
        hipFree(warmup);
        std::cout << "GPU warmup finished" << std::endl;

        unsigned long long count = 0;

        //int H_size = nonzerovertex.size();
        unsigned H_size = firstlevel.size();

        unsigned* row_offset_dev_L, * column_index_dev_L, * row_offset_dev_H, * column_index_dev_H, * non_vertex_dev, * non_vertex_dev_father;

        hipMalloc((void**)&row_offset_dev_L, (graphL.vertex_num + 1) * sizeof(unsigned));
        hipMalloc((void**)&column_index_dev_L, graphL.edge_num_after_trim * sizeof(unsigned));
        hipMalloc((void**)&row_offset_dev_H, (graphH.vertex_num + 1) * sizeof(unsigned));
        hipMalloc((void**)&column_index_dev_H, graphH.edge_num * sizeof(unsigned));
        checkCudaErrors(hipGetLastError());

        //hipMalloc((void**)&non_vertex_dev, nonzerovertex.size() * sizeof(unsigned));
        //hipMemcpy(non_vertex_dev, &nonzerovertex[0], nonzerovertex.size() * sizeof(unsigned), hipMemcpyHostToDevice);
        //********* New Stretagy ******************
        hipMalloc((void**)&non_vertex_dev, firstlevel.size() * sizeof(unsigned));
        hipMemcpy(non_vertex_dev, &firstlevel[0], firstlevel.size() * sizeof(unsigned), hipMemcpyHostToDevice);
        hipMalloc((void**)&non_vertex_dev_father, firstlevelfather.size() * sizeof(unsigned));
        hipMemcpy(non_vertex_dev_father, &firstlevelfather[0], firstlevelfather.size() * sizeof(unsigned), hipMemcpyHostToDevice);
        //*****************************************

        hipMemcpy(row_offset_dev_L, csrL.row_offset, (graphL.vertex_num + 1) * sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());
        hipMemcpy(column_index_dev_L, csrL.column_index, graphL.edge_num_after_trim * sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());

        hipMemcpy(row_offset_dev_H, csrH.row_offset, (graphH.vertex_num + 1) * sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());
        hipMemcpy(column_index_dev_H, csrH.column_index, graphH.edge_num * sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());

        unsigned * p_dev, * q_dev;
        unsigned long long * count_dev;
        unsigned* H_size_dev;
        hipMalloc((void**)&count_dev, sizeof(unsigned long long));
        hipMalloc((void**)&q_dev, sizeof(unsigned));
        hipMalloc((void**)&H_size_dev, sizeof(unsigned));
        hipMalloc((void**)&p_dev, sizeof(unsigned));

        hipMemcpy(count_dev, &count, sizeof(unsigned long long), hipMemcpyHostToDevice);
        hipMemcpy(q_dev, &q, sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(p_dev, &p, sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(H_size_dev, &H_size, sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());

        auto start = std::chrono::high_resolution_clock::now();
        findCliqueGPUNew7Dis2LevelStealWorkBatch << <BLOCKNUM, THREADNUM >> > (row_offset_dev_L, column_index_dev_L, row_offset_dev_H, column_index_dev_H, count_dev, p_dev, q_dev, H_size_dev, non_vertex_dev, non_vertex_dev_father);
        checkCudaErrors(hipGetLastError());
        hipMemcpy(&count, count_dev, sizeof(unsigned long long), hipMemcpyDeviceToHost);
        checkCudaErrors(hipGetLastError());
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> duration = end - start;
        std::cout << "\nAll time: " << duration.count() << "s" << std::endl;
        

        

        hipFree(non_vertex_dev);

        hipFree(row_offset_dev_L);
        hipFree(column_index_dev_L);
        hipFree(row_offset_dev_H);
        hipFree(column_index_dev_H);
        hipFree(count_dev);
        hipFree(p_dev);
        hipFree(q_dev);
        hipFree(H_size_dev);
        //************* New Stretagy ****************
        hipFree(non_vertex_dev_father);

        std::cout << "The number of (" << p << "," << q << ")-biclique is " << count << std::endl;
    }
    else{
        printf("Select right to construct H\n");
        readFile(graphL, false, path);
        for (int i = 0; i < graphL.vertex_num; i++) {
            //printf("%d,", graphL.vertices[i].neighbor.size());
            sort(graphL.vertices[i].neighbor.begin(), graphL.vertices[i].neighbor.end());
        }
        std::cout << "vertexNum:" << graphL.vertex_num << "; edgeNum:" << graphL.edge_num << std::endl;


        Graph graphR;
        readFile(graphR, true, path);
        for (int i = 0; i < graphR.vertex_num; i++) {
            sort(graphR.vertices[i].neighbor.begin(), graphR.vertices[i].neighbor.end());
        }
        std::cout << "vertexNum:" << graphR.vertex_num << "; edgeNum:" << graphR.edge_num << std::endl;

        TrimGraphByCoreNew(graphL, graphR, q, p);
        std::cout << "vertexNum:" << graphL.vertex_num_after_trim << "; edgeNum:" << graphL.edge_num_after_trim << std::endl;
        std::cout << "vertexNum:" << graphR.vertex_num_after_trim << "; edgeNum:" << graphR.edge_num_after_trim << std::endl;

        // reformatGraph(graphL, graphR);
		// std::cout << "Reformat Done" << std::endl;

        Graph graphH;
        Collect2Hop(graphL, graphR, graphH, p);
        std::cout << "Construct H End" << std::endl;
        // std::cout << "vertexNum:" << graphH.vertex_num << "; edgeNum:" << graphH.edge_num << std::endl;
        edgeDirectingByDegreeNew(graphH);
        std::cout << "vertexNum:" << graphH.vertex_num << "; edgeNum:" << graphH.edge_num << std::endl;

        int zero_count = 0;
        std::vector<unsigned> nonzerovertex;
        for (int i = 0; i < graphH.vertex_num; i++) {
            if (graphH.vertices[i].neighbor.size() < q - 1) {
                zero_count++;
            }
            else {
                nonzerovertex.push_back(i);
            }
        }
        std::cout << "Number of degree-zero vertex: " << zero_count << "; Others: " << nonzerovertex.size() << std::endl;
        //******** New Stretagy ************
        std::vector<unsigned> firstlevel;
        std::vector<unsigned> firstlevelfather;
        for (int i = 0; i < nonzerovertex.size(); i++) {
            for (int j = 0; j < graphH.vertices[nonzerovertex[i]].neighbor.size(); j++) {
                firstlevel.push_back(graphH.vertices[nonzerovertex[i]].neighbor[j] - 1);
                firstlevelfather.push_back(nonzerovertex[i]);
            }
        }
        std::cout << "Number of first level: " << firstlevel.size() << std::endl;
        //**********************************

        /*unsigned aa = nonzerovertex[514];
        nonzerovertex.clear();
        nonzerovertex.push_back(aa);*/
        /*for (int i = 0; i < nonzerovertex.size(); i++) {
            printf("%d,", graphH.vertices[nonzerovertex[i]].neighbor.size());
        }*/

        /*unsigned* all_nei = new unsigned[12]{};
        for (int i = 0; i < nonzerovertex.size(); i++) {
            printf("tid:%d, num:%d, binum:%d\n", i, graphH.vertices[nonzerovertex[i]].neighbor.size(), graphL.vertices[nonzerovertex[i]].neighbor.size());
            all_nei[i % 12] += graphH.vertices[nonzerovertex[i]].neighbor.size();
        }
        for (int i = 0; i < 12; i++) printf("block: %d, allNeighbor:%d\n", i, all_nei[i]);*/

        graphL.transformToCSR(csrL);
        graphH.transformToCSR(csrH);

        //warm up GPU
        int* warmup = NULL;
        hipMalloc(&warmup, sizeof(int));
        hipFree(warmup);
        std::cout << "GPU warmup finished" << std::endl;

        unsigned long long count = 0;

        //int H_size = nonzerovertex.size();
        unsigned H_size = firstlevel.size();

        unsigned* row_offset_dev_L, * column_index_dev_L, * row_offset_dev_H, * column_index_dev_H, * non_vertex_dev, * non_vertex_dev_father;

        hipMalloc((void**)&row_offset_dev_L, (graphL.vertex_num + 1) * sizeof(unsigned));
        hipMalloc((void**)&column_index_dev_L, graphL.edge_num_after_trim * sizeof(unsigned));
        hipMalloc((void**)&row_offset_dev_H, (graphH.vertex_num + 1) * sizeof(unsigned));
        hipMalloc((void**)&column_index_dev_H, graphH.edge_num * sizeof(unsigned));
        checkCudaErrors(hipGetLastError());

        //hipMalloc((void**)&non_vertex_dev, nonzerovertex.size() * sizeof(unsigned));
        //hipMemcpy(non_vertex_dev, &nonzerovertex[0], nonzerovertex.size() * sizeof(unsigned), hipMemcpyHostToDevice);
        //********* New Stretagy ******************
        hipMalloc((void**)&non_vertex_dev, firstlevel.size() * sizeof(unsigned));
        hipMemcpy(non_vertex_dev, &firstlevel[0], firstlevel.size() * sizeof(unsigned), hipMemcpyHostToDevice);
        hipMalloc((void**)&non_vertex_dev_father, firstlevelfather.size() * sizeof(unsigned));
        hipMemcpy(non_vertex_dev_father, &firstlevelfather[0], firstlevelfather.size() * sizeof(unsigned), hipMemcpyHostToDevice);
        //*****************************************

        hipMemcpy(row_offset_dev_L, csrL.row_offset, (graphL.vertex_num + 1) * sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());
        hipMemcpy(column_index_dev_L, csrL.column_index, graphL.edge_num_after_trim * sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());

        hipMemcpy(row_offset_dev_H, csrH.row_offset, (graphH.vertex_num + 1) * sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());
        hipMemcpy(column_index_dev_H, csrH.column_index, graphH.edge_num * sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());

        unsigned * p_dev, * q_dev;
        unsigned long long * count_dev;
        unsigned* H_size_dev;
        hipMalloc((void**)&count_dev, sizeof(unsigned long long));
        hipMalloc((void**)&q_dev, sizeof(unsigned));
        hipMalloc((void**)&H_size_dev, sizeof(unsigned));
        hipMalloc((void**)&p_dev, sizeof(unsigned));

        hipMemcpy(count_dev, &count, sizeof(unsigned long long), hipMemcpyHostToDevice);
        hipMemcpy(q_dev, &q, sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(p_dev, &p, sizeof(unsigned), hipMemcpyHostToDevice);
        hipMemcpy(H_size_dev, &H_size, sizeof(unsigned), hipMemcpyHostToDevice);
        checkCudaErrors(hipGetLastError());

        auto start = std::chrono::high_resolution_clock::now();
        findCliqueGPUNew7Dis2LevelStealWorkBatch << <BLOCKNUM, THREADNUM >> > (row_offset_dev_L, column_index_dev_L, row_offset_dev_H, column_index_dev_H, count_dev, q_dev, p_dev, H_size_dev, non_vertex_dev, non_vertex_dev_father);
        checkCudaErrors(hipGetLastError());
        hipMemcpy(&count, count_dev, sizeof(unsigned long long), hipMemcpyDeviceToHost);
        checkCudaErrors(hipGetLastError());
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<float> duration = end - start;
        std::cout << "\nAll time: " << duration.count() << "s" << std::endl;
        

        

        hipFree(non_vertex_dev);

        hipFree(row_offset_dev_L);
        hipFree(column_index_dev_L);
        hipFree(row_offset_dev_H);
        hipFree(column_index_dev_H);
        hipFree(count_dev);
        hipFree(p_dev);
        hipFree(q_dev);
        hipFree(H_size_dev);
        //************* New Stretagy ****************
        hipFree(non_vertex_dev_father);

        std::cout << "The number of (" << p << "," << q << ")-biclique is " << count << std::endl;
    }
	//test();
	return 0;
}
